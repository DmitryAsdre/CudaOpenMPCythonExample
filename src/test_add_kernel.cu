
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void __global__ kernel_add_one(int * a, int length) {
    int gid = threadIdx.x + blockDim.x*blockIdx.x;

    while(gid < length) {
    	a[gid] += 1;
        gid += blockDim.x*gridDim.x;
    }
}

void __global__ kernel_sum_sin(float * data, int length, float * res_sum){
    int gid = threadIdx.x + blockDim.x*blockIdx.x;

    while(gid < length){
        //for(int j = 0; j < length; j++){
        //    atomicAdd(res_sum, sin(data[gid])*cos(data[j]));
        //}
        atomicAdd(res_sum, sin(data[gid]));
        gid += blockDim.x*gridDim.x;
    }
}